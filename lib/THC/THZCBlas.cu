#include "THZCBlas.h"
#include "THZCGeneral.h"
#include "hip/hip_complex.h"

void THZCudaBlas_swap(THCState *state, long n, cx *x, long incx, cx *y, long incy)
{
  if(n == 1)
  {
    incx = 1;
    incy = 1;
  }

  if( (n <= INT_MAX) && (incx <= INT_MAX) && (incy <= INT_MAX) )
  {
    int i_n = (int)n;
    int i_incx = (int)incx;
    int i_incy = (int)incy;
    THZCublasCheck(hipblasCswap(THCState_getCurrentBlasHandle(state), i_n, (hipComplex*)x, i_incx, (hipComplex*)y, i_incy));
    return;
  }
  THError("Cublas_swap only supports n, incx and"
          " incy upto signed integer limits: %d", INT_MAX);
}

void THZCudaBlas_scal(THCState *state, long n, cx a, cx *x, long incx)
{
  if(n == 1)
    incx = 1;

  if( (n <= INT_MAX) && (incx <= INT_MAX) )
  {
    int i_n = (int)n;
    int i_incx = (int)incx;
    THZCublasCheck(hipblasCscal(THCState_getCurrentBlasHandle(state), i_n, (hipComplex*)&a, (hipComplex*)x, i_incx));
    return;
  }
  THError("Cublas_scal only supports n and incx "
          "upto signed integer limits: %d", INT_MAX);
}

void THZCudaBlas_copy(THCState *state, long n, cx *x, long incx, cx *y, long incy)
{
  if(n == 1)
  {
    incx = 1;
    incy = 1;
  }

  if( (n <= INT_MAX) && (incx <= INT_MAX) && (incy <= INT_MAX) )
  {
    int i_n = (int)n;
    int i_incx = (int)incx;
    int i_incy = (int)incy;
    THZCublasCheck(hipblasCcopy(THCState_getCurrentBlasHandle(state), i_n, (hipComplex*)x, i_incx, (hipComplex*)y, i_incy));
    return;
  }

  THError("Cublas_copy only supports n, incx and incy "
          "upto signed integer limits: %d", INT_MAX);
}

void THZCudaBlas_axpy(THCState *state, long n, cx a, cx *x, long incx, cx *y, long incy)
{
    if(n == 1)
  {
    incx = 1;
    incy = 1;
  }

  if( (n <= INT_MAX) && (incx <= INT_MAX) && (incy <= INT_MAX) )
  {
    int i_n = (int)n;
    int i_incx = (int)incx;
    int i_incy = (int)incy;
    THZCublasCheck(hipblasCaxpy(THCState_getCurrentBlasHandle(state), i_n, (hipComplex*)&a, (hipComplex*)x, i_incx, (hipComplex*)y, i_incy));
    return;
  }

  THError("Cublas_axpy only supports n, incx and incy "
          "upto signed integer limits: %d", INT_MAX);
}

cx THZCudaBlas_dot(THCState *state, long n, cx *x, long incx, cx *y, long incy)
{
  if(n == 1)
  {
    incx = 1;
    incy = 1;
  }

  if( (n <= INT_MAX) && (incx <= INT_MAX) && (incy <= INT_MAX) )
  {
    int i_n = (int)n;
    int i_incx = (int)incx;
    int i_incy = (int)incy;
    cx result;
    THZCublasCheck(hipblasCdotu(THCState_getCurrentBlasHandle(state), i_n, (hipComplex*)x, i_incx, (hipComplex*)y, i_incy, &result));
    hipDeviceSynchronize();
    return result;
  }
  THError("Cublas_dot only supports n, incx and incy "
          "upto signed integer limits: %d", INT_MAX);
  return -1;
}

/* Level 2 */
void THZCudaBlas_gemv(THCState *state, char trans, long m, long n, cx alpha, cx *a, long lda, cx *x, long incx, cx beta, cx *y, long incy)
{
  if(n == 1)
    lda = m;

  hipblasOperation_t op;
  if (trans == 't') op = HIPBLAS_OP_T;
  else if (trans == 'n') op = HIPBLAS_OP_N;
  else if (trans == 'c') op = HIPBLAS_OP_C;

  if( (m <= INT_MAX) && (n <= INT_MAX) &&
      (lda > 0) && (lda <= INT_MAX) &&
      (incx > 0) && (incx <= INT_MAX) &&
      (incy > 0) && (incy <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_lda = (int)lda;
    int i_incx = (int)incx;
    int i_incy = (int)incy;

    THZCublasCheck(hipblasCgemv(THCState_getCurrentBlasHandle(state), op, i_m, i_n, (hipComplex*)&alpha, (hipComplex*)a, i_lda, (hipComplex*)x, i_incx, (hipComplex*)&beta, (hipComplex*)y, i_incy));
    return;
  }
  THError("Cublas_gemv only supports m, n, lda, incx, incy"
          "in the range 0 < [val] <= %d", INT_MAX);
}

void THZCudaBlas_ger(THCState *state, long m, long n, cx alpha, cx *x, long incx, cx *y, long incy, cx *a, long lda)
{
  if(n == 1)
    lda = m;

  if( (m <= INT_MAX) && (n <= INT_MAX) && (lda <= INT_MAX)  && (incx <= INT_MAX) && (incy <= INT_MAX) )
    {
      int i_m = (int)m;
      int i_n = (int)n;
      int i_lda = (int)lda;
      int i_incx = (int)incx;
      int i_incy = (int)incy;

      THZCublasCheck(hipblasCgeru(THCState_getCurrentBlasHandle(state), i_m, i_n, (hipComplex*)&alpha, (hipComplex*)x, i_incx, (hipComplex*)y, i_incy, (hipComplex*)a, i_lda));
      return;
    }
  THError("Cublas_ger only supports m, n, lda, incx, incy"
          "with the bound [val] <= %d", INT_MAX);
}

hipblasOperation_t convertTransToCublasOperation(char trans) {
  if (trans == 't') return HIPBLAS_OP_T;
  else if (trans == 'n') return HIPBLAS_OP_N;
  else if (trans == 'c') return HIPBLAS_OP_C;
  else {
    THError("trans must be one of: t, n, c");
    return HIPBLAS_OP_T;
  }
}

void adjustLd(char transa, char transb, long m, long n, long k, long *lda, long *ldb, long *ldc)
{
  int transa_ = ((transa == 't') || (transa == 'T'));
  int transb_ = ((transb == 't') || (transb == 'T'));

  if(n == 1)
    *ldc = m;

  if(transa_)
  {
    if(m == 1)
      *lda = k;
  }
  else
  {
    if(k == 1)
      *lda = m;
  }

  if(transb_)
  {
    if(k == 1)
      *ldb = n;
  }
  else
  {
    if(n == 1)
      *ldb = k;
  }
}

/* Level 3 */
void THZCudaBlas_gemm(THCState *state, char transa, char transb, long m, long n, long k, cx alpha, cx *a, long lda, cx *b, long ldb, cx beta, cx *c, long ldc)
{
  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  if( (m <= INT_MAX) && (n <= INT_MAX) && (k <= INT_MAX) && (lda <= INT_MAX)  && (ldb <= INT_MAX) && (ldc <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_k = (int)k;
    int i_lda = (int)lda;
    int i_ldb = (int)ldb;
    int i_ldc = (int)ldc;

    THZCublasCheck(hipblasCgemm(THCState_getCurrentBlasHandle(state), opa, opb, i_m, i_n, i_k, (hipComplex*)&alpha, (hipComplex*)a, i_lda, (hipComplex*)b, i_ldb, (hipComplex*)&beta, (hipComplex*)c, i_ldc));
    return;
  }
  THError("Cublas_gemm only supports m, n, k, lda, ldb, ldc"
          "with the bound [val] <= %d", INT_MAX);
}

void THZCudaBlas_gemmBatched(THCState *state, char transa, char transb, long m, long n, long k,
                            cx alpha, const cx *a[], long lda, const cx *b[], long ldb,
                            cx* beta, cx *c[], long ldc, long batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )
  {
    THError("Cublas_gemm only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  THZCublasCheck(hipblasCgemmBatched(THCState_getCurrentBlasHandle(state),
                                   opa, opb, (int)m, (int)n, (int)k,
                                   (hipComplex*)&alpha, (const hipComplex**)a, (int)lda, (const hipComplex**)b, (int)ldb, (hipComplex*)&beta, (const hipComplex**)c, (int)ldc,
                                   (int)batchCount));
}
