#include "THZCTensorMath.h"
#include "THZCGeneral.h"
#include "THZCGeneral.cuh"
#include "THZCBlas.h"
#include "THZCTensorCopy.h"
#include "THZCApply.cuh"
#include "THZCReduce.cuh"

// #include <thrust/complex.h>
// typedef thrust::complex<float> ccx;

// ccx toCcx(cx val) {
// 	return ccx(crealf(val), cimagf(val));
// }
struct TensorFillOp {
	TensorFillOp(ccx v) :
			val(v) {
	}
	__device__ __forceinline__ void operator()(ccx* v) {
		*v = val;
	}

	const ccx val;
};
struct TensorFillReOp {
	TensorFillReOp(float v) :
			val(v) {
	}
	__device__ __forceinline__ void operator()(ccx* v) {
		*v = ccx(val, v->imag());
	}

	const float val;
};
struct TensorFillImOp {
	TensorFillImOp(float v) :
			val(v) {
	}
	__device__ __forceinline__ void operator()(ccx* v) {
		*v = ccx(v->real(), val);
	}

	const float val;
};
struct TensorAddConstantOp {
  TensorAddConstantOp(ccx v) : val(v) {}
  __device__ __forceinline__ void operator()(ccx* out, ccx* in) {
    ccx *o = (ccx*)out;
    ccx *i = (ccx*)in;
    *o = *i + val;
  }

  __device__ __forceinline__ void operator()(ccx* v) {
    ccx *vo = (ccx*)v;
    *vo += val;
  }

  const ccx val;
};

void THZCudaTensor_add(THCState *state, THZCudaTensor *self_, THZCudaTensor *src_, cx value)
{
  THAssert(THZCudaTensor_checkGPU(state, 2, self_, src_));
  if (self_ == src_) {
		// printf("path1\n");
    if (!THZCudaTensor_pointwiseApply1(state, self_, TensorAddConstantOp(toCcx(value)))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THZCudaTensor_resizeAs(state, self_, src_);
		// printf("path2\n");
    if (!THZCudaTensor_pointwiseApply2(state, self_, src_, TensorAddConstantOp(toCcx(value)))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THZCudaCheck(hipGetLastError());
}

struct TensorMulConstantOp {
  TensorMulConstantOp(ccx v) : val(v) {}
  __device__ __forceinline__ void operator()(ccx* o, ccx* i) {
    *o = *i * val;
  }

  __device__ __forceinline__ void operator()(ccx* v) {
    *v *= val;
  }

  const ccx val;
};

void THZCudaTensor_fill(THCState* state, THZCudaTensor *self_, cx value) {
	THAssert(THZCudaTensor_checkGPU(state, 1, self_));
	if (!THZCudaTensor_pointwiseApply1(state, self_, TensorFillOp(toCcx(value)))) {
		THArgCheck(false, 1, CUTORCH_DIM_WARNING);
	}

	THZCudaCheck(hipGetLastError());
}

void THZCudaTensor_fillim(THCState* state, THZCudaTensor *self_, float value) {
	THAssert(THZCudaTensor_checkGPU(state, 1, self_));
	if (!THZCudaTensor_pointwiseApply1(state, self_, TensorFillImOp(value))) {
		THArgCheck(false, 1, CUTORCH_DIM_WARNING);
	}

	THZCudaCheck(hipGetLastError());
}

void THZCudaTensor_fillre(THCState* state, THZCudaTensor *self_, float value) {
	THAssert(THZCudaTensor_checkGPU(state, 1, self_));
	if (!THZCudaTensor_pointwiseApply1(state, self_, TensorFillReOp(value))) {
		THArgCheck(false, 1, CUTORCH_DIM_WARNING);
	}

	THZCudaCheck(hipGetLastError());
}
void THZCudaTensor_ones(THCState *state, THZCudaTensor *r_,
		THLongStorage *size) {
	THAssert(THZCudaTensor_checkGPU(state, 1, r_));
	THZCudaTensor_resize(state, r_, size, NULL);
	THZCudaTensor_fill(state, r_, 1);
}
void THZCudaTensor_zero(THCState *state, THZCudaTensor *self_) {
	THAssert(THZCudaTensor_checkGPU(state, 1, self_));
	if (THZCudaTensor_isContiguous(state, self_)) {
		THZCudaCheck(
				hipMemsetAsync(THZCudaTensor_data(state, self_), 0, sizeof(cx) * THZCudaTensor_nElement(state, self_), THCState_getCurrentStream(state)));
	} else {
		if (!THZCudaTensor_pointwiseApply1(state, self_, TensorFillOp(0))) {
			THArgCheck(false, 1, CUTORCH_DIM_WARNING);
		}
	}

	THZCudaCheck(hipGetLastError());
}

void THZCudaTensor_zeros(THCState *state, THZCudaTensor *r_,
		THLongStorage *size) {
	THAssert(THZCudaTensor_checkGPU(state, 1, r_));
	THZCudaTensor_resize(state, r_, size, NULL);
	THZCudaTensor_zero(state, r_);
}
void THZCudaTensor_mul(THCState *state, THZCudaTensor *self_, THZCudaTensor *src_, cx value)
{
  THAssert(THZCudaTensor_checkGPU(state, 2, self_, src_));
  if (self_ == src_) {
    if (!THZCudaTensor_pointwiseApply1(state, self_, TensorMulConstantOp(toCcx(value)))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THZCudaTensor_resizeAs(state, self_, src_);

    if (!THZCudaTensor_pointwiseApply2(state, self_, src_, TensorMulConstantOp(toCcx(value)))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THZCudaCheck(hipGetLastError());
}

void THZCudaTensor_div(THCState* state, THZCudaTensor *self_, THZCudaTensor *src_, cx value)
{
  THAssert(THZCudaTensor_checkGPU(state, 2, self_, src_));
  THArgCheck(value != 0.0f, 3, "divide by zero");

  if (self_ == src_) {
    if (!THZCudaTensor_pointwiseApply1(state, self_, TensorMulConstantOp(1.0f / toCcx(value)))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THZCudaTensor_resizeAs(state, self_, src_);

    if (!THZCudaTensor_pointwiseApply2(state, self_, src_, TensorMulConstantOp(1.0f / toCcx(value)))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THZCudaCheck(hipGetLastError());
}

template <int Upper>
struct TensorTriOp {
  TensorTriOp(cx *start_, long stride0_, long stride1_, long k_)
    : start(start_), stride0(stride0_), stride1(stride1_), k(k_) {}

  __device__ __forceinline__ int mask(ccx *in) {
    ptrdiff_t n = in - (ccx*)start;
    long row, col;
    if (stride0 > stride1)
    {
      row = (long) (n / stride0);
      col = (long) ((n % stride0) / stride1);
    }
    else
    {
      row = (long) ((n % stride1) / stride0);
      col = (long) (n / stride1);
    }

    return Upper ? (col - row >= k) : (col - row <= k);
  }

  __device__ __forceinline__ void operator()(ccx* out, ccx* in) {
    *out = mask(in) ? *in : 0;
  }

  __device__ __forceinline__ void operator()(ccx* v) {
    if (!mask(v))
      *v = 0;
  }

  const cx *start;
  const long stride0, stride1, k;
};

void THZCudaTensor_tril(THCState *state, THZCudaTensor *self_, THZCudaTensor *src_, long k)
{
  THAssert(THZCudaTensor_checkGPU(state, 2, self_, src_));
  THArgCheck(src_->nDimension == 2, 1, "expected a matrix");

  THZCudaTensor *src = src_;
  if (self_ == src_)
    src = THZCudaTensor_newContiguous(state, src_);

  long stride0 = src->stride[0];
  long stride1 = src->stride[1];
  cx *start = THZCudaTensor_data(state, src) + src->storageOffset;

  TensorTriOp<0> op(start, stride0, stride1, k);

  if (self_ == src_) {
    if (!THZCudaTensor_pointwiseApply1(state, src, op)) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THZCudaTensor_resizeAs(state, self_, src);

    if (!THZCudaTensor_pointwiseApply2(state, self_, src, op)) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  if (self_ == src_)
    THZCudaTensor_freeCopyTo(state, src, src_);

  THZCudaCheck(hipGetLastError());
}

void THZCudaTensor_triu(THCState *state, THZCudaTensor *self_, THZCudaTensor *src_, long k)
{
  THAssert(THZCudaTensor_checkGPU(state, 2, self_, src_));
  THArgCheck(src_->nDimension == 2, 1, "expected a matrix");

  THZCudaTensor *src = src_;
  if (self_ == src_)
    src = THZCudaTensor_newContiguous(state, src_);

  long stride0 = src->stride[0];
  long stride1 = src->stride[1];
  cx *start = THZCudaTensor_data(state, src) + src->storageOffset;

  TensorTriOp<1> op(start, stride0, stride1, k);

  if (self_ == src_) {
    if (!THZCudaTensor_pointwiseApply1(state, src, op)) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THZCudaTensor_resizeAs(state, self_, src);

    if (!THZCudaTensor_pointwiseApply2(state, self_, src, op)) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  if (self_ == src_)
    THZCudaTensor_freeCopyTo(state, src, src_);

  THZCudaCheck(hipGetLastError());
}
