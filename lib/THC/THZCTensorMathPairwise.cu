#include "THZCTensorMath.h"
#include "THZCGeneral.h"
#include "THZCGeneral.cuh"
#include "THZCBlas.h"
#include "THZCTensorCopy.h"
#include "THZCApply.cuh"
#include "THZCReduce.cuh"

// #include <thrust/complex.h>
// typedef thrust::complex<float> ccx;

// ccx toCcx(cx val) {
// 	return ccx(crealf(val), cimagf(val));
// }

struct TensorAddConstantOp {
  TensorAddConstantOp(ccx v) : val(v) {}
  __device__ __forceinline__ void operator()(ccx* out, ccx* in) {
    ccx *o = (ccx*)out;
    ccx *i = (ccx*)in;
    *o = *i + val;
  }

  __device__ __forceinline__ void operator()(ccx* v) {
    ccx *vo = (ccx*)v;
    *vo += val;
  }

  const ccx val;
};

void THZCudaTensor_add(THCState *state, THZCudaTensor *self_, THZCudaTensor *src_, cx value)
{
  THAssert(THZCudaTensor_checkGPU(state, 2, self_, src_));
  if (self_ == src_) {
    if (!THZCudaTensor_pointwiseApply1(state, self_, TensorAddConstantOp(toCcx(value)))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THZCudaTensor_resizeAs(state, self_, src_);

    if (!THZCudaTensor_pointwiseApply2(state, self_, src_, TensorAddConstantOp(toCcx(value)))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THZCudaCheck(hipGetLastError());
}

struct TensorMulConstantOp {
  TensorMulConstantOp(ccx v) : val(v) {}
  __device__ __forceinline__ void operator()(ccx* o, ccx* i) {
    *o = *i * val;
  }

  __device__ __forceinline__ void operator()(ccx* v) {
    *v *= val;
  }

  const ccx val;
};

void THZCudaTensor_mul(THCState *state, THZCudaTensor *self_, THZCudaTensor *src_, cx value)
{
  THAssert(THZCudaTensor_checkGPU(state, 2, self_, src_));
  if (self_ == src_) {
    if (!THZCudaTensor_pointwiseApply1(state, self_, TensorMulConstantOp(toCcx(value)))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THZCudaTensor_resizeAs(state, self_, src_);

    if (!THZCudaTensor_pointwiseApply2(state, self_, src_, TensorMulConstantOp(toCcx(value)))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THZCudaCheck(hipGetLastError());
}

void THZCudaTensor_div(THCState* state, THZCudaTensor *self_, THZCudaTensor *src_, cx value)
{
  THAssert(THZCudaTensor_checkGPU(state, 2, self_, src_));
  THArgCheck(value != 0.0f, 3, "divide by zero");

  if (self_ == src_) {
    if (!THZCudaTensor_pointwiseApply1(state, self_, TensorMulConstantOp(1.0f / toCcx(value)))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THZCudaTensor_resizeAs(state, self_, src_);

    if (!THZCudaTensor_pointwiseApply2(state, self_, src_, TensorMulConstantOp(1.0f / toCcx(value)))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THZCudaCheck(hipGetLastError());
}

template <int Upper>
struct TensorTriOp {
  TensorTriOp(cx *start_, long stride0_, long stride1_, long k_)
    : start(start_), stride0(stride0_), stride1(stride1_), k(k_) {}

  __device__ __forceinline__ int mask(ccx *in) {
    ptrdiff_t n = in - (ccx*)start;
    long row, col;
    if (stride0 > stride1)
    {
      row = (long) (n / stride0);
      col = (long) ((n % stride0) / stride1);
    }
    else
    {
      row = (long) ((n % stride1) / stride0);
      col = (long) (n / stride1);
    }

    return Upper ? (col - row >= k) : (col - row <= k);
  }

  __device__ __forceinline__ void operator()(ccx* out, ccx* in) {
    *out = mask(in) ? *in : 0;
  }

  __device__ __forceinline__ void operator()(ccx* v) {
    if (!mask(v))
      *v = 0;
  }

  const cx *start;
  const long stride0, stride1, k;
};

void THZCudaTensor_tril(THCState *state, THZCudaTensor *self_, THZCudaTensor *src_, long k)
{
  THAssert(THZCudaTensor_checkGPU(state, 2, self_, src_));
  THArgCheck(src_->nDimension == 2, 1, "expected a matrix");

  THZCudaTensor *src = src_;
  if (self_ == src_)
    src = THZCudaTensor_newContiguous(state, src_);

  long stride0 = src->stride[0];
  long stride1 = src->stride[1];
  cx *start = THZCudaTensor_data(state, src) + src->storageOffset;

  TensorTriOp<0> op(start, stride0, stride1, k);

  if (self_ == src_) {
    if (!THZCudaTensor_pointwiseApply1(state, src, op)) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THZCudaTensor_resizeAs(state, self_, src);

    if (!THZCudaTensor_pointwiseApply2(state, self_, src, op)) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  if (self_ == src_)
    THZCudaTensor_freeCopyTo(state, src, src_);

  THZCudaCheck(hipGetLastError());
}

void THZCudaTensor_triu(THCState *state, THZCudaTensor *self_, THZCudaTensor *src_, long k)
{
  THAssert(THZCudaTensor_checkGPU(state, 2, self_, src_));
  THArgCheck(src_->nDimension == 2, 1, "expected a matrix");

  THZCudaTensor *src = src_;
  if (self_ == src_)
    src = THZCudaTensor_newContiguous(state, src_);

  long stride0 = src->stride[0];
  long stride1 = src->stride[1];
  cx *start = THZCudaTensor_data(state, src) + src->storageOffset;

  TensorTriOp<1> op(start, stride0, stride1, k);

  if (self_ == src_) {
    if (!THZCudaTensor_pointwiseApply1(state, src, op)) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THZCudaTensor_resizeAs(state, self_, src);

    if (!THZCudaTensor_pointwiseApply2(state, self_, src, op)) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  if (self_ == src_)
    THZCudaTensor_freeCopyTo(state, src, src_);

  THZCudaCheck(hipGetLastError());
}
