#include "THZCTensorMath.h"
#include "FFT.h"
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include "hip/hip_runtime.h"
#include ""
#include "THZCGeneral.h"
#include "THZCGeneral.cuh"

// #include "arrayfire.h"

#ifdef _CUFFT_H_
// cuFFT API errors
static const char *_cudaGetErrorEnum(hipfftResult error)
{
	switch (error)
	{
		case HIPFFT_SUCCESS:
		return "HIPFFT_SUCCESS";

		case HIPFFT_INVALID_PLAN:
		return "HIPFFT_INVALID_PLAN";

		case HIPFFT_ALLOC_FAILED:
		return "HIPFFT_ALLOC_FAILED";

		case HIPFFT_INVALID_TYPE:
		return "HIPFFT_INVALID_TYPE";

		case HIPFFT_INVALID_VALUE:
		return "HIPFFT_INVALID_VALUE";

		case HIPFFT_INTERNAL_ERROR:
		return "HIPFFT_INTERNAL_ERROR";

		case HIPFFT_EXEC_FAILED:
		return "HIPFFT_EXEC_FAILED";

		case HIPFFT_SETUP_FAILED:
		return "HIPFFT_SETUP_FAILED";

		case HIPFFT_INVALID_SIZE:
		return "HIPFFT_INVALID_SIZE";

		case HIPFFT_UNALIGNED_DATA:
		return "HIPFFT_UNALIGNED_DATA";
	}

	return "<unknown>";
}
#endif

void cufftShift_2D_kernel(ccx* d, int nx, int ny)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // // 2D Slice & 1D Line
    // int sLine = N;
    // int sSlice = N * N;
		//
    // // Transformations Equations
    // int sEq1 = (sSlice + sLine) / 2;
    // int sEq2 = (sSlice - sLine) / 2;
		//
    // // Thread Index (1D)
    // int xThreadIdx = threadIdx.x;
    // int yThreadIdx = threadIdx.y;
		//
    // // Block Width & Height
    // int blockWidth = blockDim.x;
    // int blockHeight = blockDim.y;
		//
    // // Thread Index (2D)
    // int xIndex = blockIdx.x * blockWidth + xThreadIdx;
    // int yIndex = blockIdx.y * blockHeight + yThreadIdx;
		//
    // // Thread Index Converted into 1D Index
    // int index = (yIndex * N) + xIndex;
		//
    // T regTemp;
		//
    // if (xIndex < N / 2)
    // {
    //     if (yIndex < N / 2)
    //     {
    //         regTemp = data[index];
		//
    //         // First Quad
    //         data[index] = data[index + sEq1];
		//
    //         // Third Quad
    //         data[index + sEq1] = regTemp;
    //     }
    // }
    // else
    // {
    //     if (yIndex < N / 2)
    //     {
    //         regTemp = data[index];
		//
    //         // Second Quad
    //         data[index] = data[index + sEq2];
		//
    //         // Fourth Quad
    //         data[index + sEq2] = regTemp;
    //     }
    // }
}

#define cufftSafeCall(err)      __cufftSafeCall(err, __FILE__, __LINE__)
inline void __cufftSafeCall(hipfftResult err, const char *file, const int line) {
	if (HIPFFT_SUCCESS != err) {
		fprintf(stderr, "CUFFT error in file '%s', line %d\n %s\nerror %d: %s\nterminating!\n",
		__FILE__, __LINE__, err, _cudaGetErrorEnum(err));
		hipDeviceReset();
		// assert(0);
	}
}
void THZCudaTensor_fft(THCState *state, THZCudaTensor *self, THZCudaTensor *result, int direction) {
	int ndim = THZCudaTensor_nDimension(state, self);
	int batch = 1;
	int fft_dims[4];
	for (int i = 0; i < ndim; i++) {
		fft_dims[i] = (int) THZCudaTensor_size(state, self, i);
	}
	hipfftHandle plan;
	cufftSafeCall(hipfftPlanMany(&plan, ndim, fft_dims, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2C, batch));
	cufftSafeCall(hipfftSetStream(plan, THCState_getCurrentStream(state)));
	cufftSafeCall(hipfftExecC2C(plan, (hipfftComplex *)THZCudaTensor_data(state, self), (hipfftComplex *)THZCudaTensor_data(state, result), direction));
	hipfftDestroy(plan);
}
void THZCudaTensor_fftBatched(THCState *state, THZCudaTensor *self, THZCudaTensor *result, int direction) {
	int ndim = THZCudaTensor_nDimension(state, self);
	int batch = THZCudaTensor_size(state, self, 0);
	int fft_dims[3];
	for (int i = 1; i < ndim; i++) {
		fft_dims[i - 1] = (int) THZCudaTensor_size(state, self, i);
	}
	hipfftHandle plan;
	cufftSafeCall(hipfftPlanMany(&plan, ndim - 1, fft_dims, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2C, batch));
	cufftSafeCall(hipfftSetStream(plan, THCState_getCurrentStream(state)));
	cufftSafeCall(hipfftExecC2C(plan, (hipfftComplex *)THZCudaTensor_data(state, self), (hipfftComplex *)THZCudaTensor_data(state, result), direction));
	hipfftDestroy(plan);
}
void THZCudaTensor_fftInplace(THCState *s, THZCudaTensor *self) {
	THZCudaTensor_fft(s, self, self);
}
void THZCudaTensor_fft(THCState *state, THZCudaTensor *self, THZCudaTensor *result) {
	THZCudaTensor_fft(state, self, result, HIPFFT_FORWARD);
}
void THZCudaTensor_fftBatchedInplace(THCState *state, THZCudaTensor *self) {
	THZCudaTensor_fftBatched(state, self, self);
}
void THZCudaTensor_fftBatched(THCState *state, THZCudaTensor *self, THZCudaTensor *result) {
	THZCudaTensor_fftBatched(state, self, result, HIPFFT_FORWARD);
}

void THZCudaTensor_ifftInplace(THCState *state, THZCudaTensor *self) {
	THZCudaTensor_ifft(state, self, self);
}
void THZCudaTensor_ifft(THCState *state, THZCudaTensor *self, THZCudaTensor *result) {
	THZCudaTensor_ifftU(state, self, result);
	THZCudaTensor_mul(state, result, result, 1 / THZCudaTensor_nElement(state, result));
}
void THZCudaTensor_ifftBatchedInplace(THCState *state, THZCudaTensor *self) {
	THZCudaTensor_ifftBatched(state, self, self);
}
void THZCudaTensor_ifftBatched(THCState *state, THZCudaTensor *self, THZCudaTensor *result) {
	THZCudaTensor_ifftBatchedU(state, self, result);
	THZCudaTensor_mul(state, result, result, 1 / THZCudaTensor_nElement(state, result));
}

void THZCudaTensor_ifftUInplace(THCState *state, THZCudaTensor *self) {
	THZCudaTensor_ifftU(state, self, self);
}
void THZCudaTensor_ifftU(THCState *state, THZCudaTensor *self, THZCudaTensor *result) {
	THZCudaTensor_fft(state, self, result, HIPFFT_BACKWARD);
}
void THZCudaTensor_ifftBatchedUInplace(THCState *state, THZCudaTensor *self) {
	THZCudaTensor_ifftBatchedU(state, self, self);
}
void THZCudaTensor_ifftBatchedU(THCState *state, THZCudaTensor *self, THZCudaTensor *result) {
	THZCudaTensor_fftBatched(state, self, result, HIPFFT_BACKWARD);
}

void THZCudaTensor_fftShiftInplace(THCState *state, THZCudaTensor *self) {

}
void THZCudaTensor_fftShift(THCState *state, THZCudaTensor *self, THZCudaTensor *result) {
	// int ndim = THZCudaTensor_nDimension(s, self);
	// int dims[4];
	// for (int i = 0; i < ndim; i++) {
	// 	dims[i] = (int) THZCudaTensor_size(s, self, i);
	// }
	// dim4 dims((const unsigned) ndim, (const long long *) dims);
	// array a(dims, (af::cfloat*)THZCudaTensor_data(state, self),af::afDevice);
	// array out = af::shift(a, a.dims(0)/2, a.dims(1)/2);

}
void THZCudaTensor_ifftShiftInplace(THCState *state, THZCudaTensor *self) {

}
void THZCudaTensor_ifftShift(THCState *state, THZCudaTensor *self, THZCudaTensor *result) {
	// int ndim = THZCudaTensor_nDimension(s, self);
	// int dims[4];
	// for (int i = 0; i < ndim; i++) {
	// 	dims[i] = (int) THZCudaTensor_size(s, self, i);
	// }
	// dim4 dims((const unsigned) ndim, (const long long *) dims);
	// array a(dims, (af::cfloat*)THZCudaTensor_data(state, self),af::afDevice);
	// array out = af::shift(wave, (a.dims(0)+1)/2, (a.dims(1)+1)/2);
}
