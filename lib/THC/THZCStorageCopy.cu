#include "THZCStorageCopy.h"
#include "THZCGeneral.h"


void THZCudaStorage_rawCopy(THCState *state, THZCudaStorage *self, cx *src)
{
  THZCudaCheck(hipMemcpyAsync(self->data, src, self->size * sizeof(cx), hipMemcpyDeviceToDevice, THCState_getCurrentStream(state)));
}

void THZCudaStorage_copy(THCState *state, THZCudaStorage *self, THZCudaStorage *src)
{
  THArgCheck(self->size == src->size, 2, "size does not match");
  THZCudaCheck(hipMemcpyAsync(self->data, src->data, self->size * sizeof(cx), hipMemcpyDeviceToDevice, THCState_getCurrentStream(state)));
}

void THZCudaStorage_copyZCuda(THCState *state, THZCudaStorage *self, THZCudaStorage *src)
{
  THArgCheck(self->size == src->size, 2, "size does not match");
  THZCudaCheck(hipMemcpyAsync(self->data, src->data, self->size * sizeof(cx), hipMemcpyDeviceToDevice, THCState_getCurrentStream(state)));
}
