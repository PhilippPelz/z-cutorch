#include "THZCStorageCopy.h"
#include "THZCGeneral.h"
#include "hip/hip_complex.h"

void THZCudaStorage_rawCopy(THCState *state, THZCudaStorage *self, hipComplex *src)
{
  THZCudaCheck(hipMemcpyAsync(self->data, src, self->size * sizeof(hipComplex), hipMemcpyDeviceToDevice, THCState_getCurrentStream(state)));
}

void THZCudaStorage_copy(THCState *state, THZCudaStorage *self, THZCudaStorage *src)
{
  THArgCheck(self->size == src->size, 2, "size does not match");
  THZCudaCheck(hipMemcpyAsync(self->data, src->data, self->size * sizeof(hipComplex), hipMemcpyDeviceToDevice, THCState_getCurrentStream(state)));
}

void THZCudaStorage_copyZCuda(THCState *state, THZCudaStorage *self, THZCudaStorage *src)
{
  THArgCheck(self->size == src->size, 2, "size does not match");
  THZCudaCheck(hipMemcpyAsync(self->data, src->data, self->size * sizeof(hipComplex), hipMemcpyDeviceToDevice, THCState_getCurrentStream(state)));
}
