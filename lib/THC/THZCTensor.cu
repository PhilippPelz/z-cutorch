#include "THZCTensor.h"

hipTextureObject_t THZCudaTensor_getTextureObject(THCState *state, THZCudaTensor *self)
{
  THAssert(THZCudaTensor_checkGPU(state, 1, self));
  hipTextureObject_t texObj;
  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeLinear;
  resDesc.res.linear.devPtr = THZCudaTensor_dataCu(state, self);
  resDesc.res.linear.sizeInBytes = THZCudaTensor_nElement(state, self) * 4;
  resDesc.res.linear.desc = hipCreateChannelDesc(32, 0, 0, 0,
                                                  hipChannelFormatKindFloat);
  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);
  hipError_t errcode = hipGetLastError();
  if(errcode != hipSuccess) {
    if (THZCudaTensor_nElement(state, self) > 2>>27)
      THError("Failed to create texture object, "
              "nElement:%ld exceeds 27-bit addressing required for tex1Dfetch. Cuda Error: %s",
              THZCudaTensor_nElement(state, self), hipGetErrorString(errcode));
    else
      THError("Failed to create texture object: %s", hipGetErrorString(errcode));
  }
  return texObj;
}

THZC_API int THZCudaTensor_getDevice(THCState* state, const THZCudaTensor* thc) {
  if (!thc->storage) return -1;
  hipPointerAttribute_t attr;
  THZCudaCheck(hipPointerGetAttributes(&attr, thc->storage->data));
  return attr.device;
}
