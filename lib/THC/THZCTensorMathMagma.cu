#include "hip/hip_runtime.h"
#include "THZCGeneral.h"
#include "THZCTensorMath.h"
#include "THZCTensorCopy.h"
#include <algorithm>

#ifdef USE_MAGMA
#include <magma.h>
#endif

#ifndef DIVUP
#define DIVUP(x, y) (((x) + (y) - 1) / (y))
#endif

#define NoMagma(name) "No CUDA implementation of '" #name "'. Install MAGMA and rebuild cutorch (http://icl.cs.utk.edu/magma/)"

void THZCMagma_init(THCState *state)
{
#ifdef USE_MAGMA
  magma_init();
#endif
}

#ifdef USE_MAGMA
static inline float* th_magma_smalloc_pinned(size_t n)
{
  float* ptr;
  if (MAGMA_SUCCESS != magma_smalloc_pinned(&ptr, n))
    THError("$ Torch: not enough memory: you tried to allocate %dGB. Buy new RAM!", n/268435456);
  return ptr;
}

static inline int* th_magma_imalloc_pinned(size_t n)
{
  int* ptr;
  if (MAGMA_SUCCESS != magma_imalloc_pinned(&ptr, n))
    THError("$ Torch: not enough memory: you tried to allocate %dGB. Buy new RAM!", n/268435456);
  return ptr;
}

static void THZCudaTensor_copyArray1d(THCState *state, THZCudaTensor *self, float *src, int k)
{
  long size[1] = { k };
  long stride[1] = { 1 };
  THZCudaTensor_rawResize(state, self, 1, size, stride);
  size_t len = k * sizeof(float);
  THZCudaCheck(hipMemcpy(self->storage->data + self->storageOffset, src, len, hipMemcpyHostToDevice));
}

static void THZCudaTensor_copyArray2d(THCState *state, THZCudaTensor *self, float *src, int m, int n)
{
  long size[2] = { m, n };
  long stride[2] = { 1, m };
  THZCudaTensor_rawResize(state, self, 2, size, stride);
  size_t len = m * n * sizeof(float);
  THZCudaCheck(hipMemcpy(self->storage->data + self->storageOffset, src, len, hipMemcpyHostToDevice));
}

static void THZCudaTensor_copyTensor2d(THCState *state, float *dst, THZCudaTensor *self)
{
  THAssert(self->nDimension == 2);
  size_t len = THZCudaTensor_nElement(state, self)*sizeof(float);
  THZCudaTensor *temp = THZCudaTensor_newTranspose(state, self, 0, 1);
  THZCudaTensor *selfc = THZCudaTensor_newContiguous(state, temp);
  THZCudaCheck(hipMemcpy(dst, selfc->storage->data + selfc->storageOffset, len, hipMemcpyDeviceToHost));
  THZCudaTensor_free(state, temp);
  THZCudaTensor_free(state, selfc);
}

static THZCudaTensor* THZCudaTensor_newColumnMajor(THCState *state, THZCudaTensor *self, THZCudaTensor *src)
{
  THAssert(src->nDimension == 2);
  if (self == src && self->stride[0] == 1 && self->stride[1] == self->size[0])
  {
    THZCudaTensor_retain(state, self);
    return self;
  }

  if (self == src)
    self = THZCudaTensor_new(state);
  else
    THZCudaTensor_retain(state, self);

  long size[2] = { src->size[0], src->size[1] };
  long stride[2] = { 1, src->size[0] };

  THZCudaTensor_rawResize(state, self, 2, size, stride);
  THZCudaTensor_copy(state, self, src);
  return self;
}
#endif

void THZCudaTensor_gesv(THCState *state, THZCudaTensor *rb_, THZCudaTensor *ra_, THZCudaTensor *b_, THZCudaTensor *a_)
{
#ifdef USE_MAGMA
  THArgCheck(a_->nDimension == 2, 1, "A should be 2 dimensional");
  THArgCheck(b_->nDimension == 2, 2, "b should be 2 dimensional");
  THArgCheck(a_->size[0] == a_->size[1], 1, "A should be square");
  THArgCheck(b_->size[0] == a_->size[0], 2, "A,b size incompatible");

  int n = a_->size[0];
  int nrhs = b_->size[1];

  THZCudaTensor *a = THZCudaTensor_newColumnMajor(state, ra_, a_);
  THZCudaTensor *b = THZCudaTensor_newColumnMajor(state, rb_, b_);
  float *a_data = THZCudaTensor_data(state, a);
  float *b_data = THZCudaTensor_data(state, b);

  int *ipiv = th_magma_imalloc_pinned(n);

  int info;
  magma_sgesv_gpu(n, nrhs, a_data, n, ipiv, b_data, n, &info);

  if (info < 0)
    THError("MAGMA gesv : Argument %d : illegal value", -info);
  else if (info > 0)
    THError("MAGMA gesv : U(%d,%d) is zero, singular U.", info, info);

  magma_free_pinned(ipiv);
  THZCudaTensor_freeCopyTo(state, a, ra_);
  THZCudaTensor_freeCopyTo(state, b, rb_);
#else
  THError(NoMagma(gesv));
#endif
}

void THZCudaTensor_gels(THCState *state, THZCudaTensor *rb_, THZCudaTensor *ra_, THZCudaTensor *b_, THZCudaTensor *a_)
{
#ifdef USE_MAGMA
  THArgCheck(a_->nDimension == 2, 1, "A should be 2 dimensional");
  THArgCheck(b_->nDimension == 2, 1, "b should be 2 dimensional");
  THArgCheck(a_->size[0] == b_->size[0], 2, "size incompatible A,b");
  THArgCheck(a_->size[0] >= a_->size[1], 2, "A should have m >= n");

  THZCudaTensor *a = THZCudaTensor_newColumnMajor(state, ra_, a_);
  THZCudaTensor *b = THZCudaTensor_newColumnMajor(state, rb_, b_);
  float *a_data = THZCudaTensor_data(state, a);
  float *b_data = THZCudaTensor_data(state, b);

  int m = a->size[0];
  int n = a->size[1];
  int nrhs = b->size[1];
  float wkopt;

  int info;
  magma_sgels_gpu(MagmaNoTrans, m, n, nrhs, a_data, m, b_data, m, &wkopt, -1, &info);

  float *hwork = th_magma_smalloc_pinned((size_t)wkopt);
  magma_sgels_gpu(MagmaNoTrans, m, n, nrhs, a_data, m, b_data, m, hwork, (int)wkopt, &info);
  magma_free_pinned(hwork);

  if (info != 0)
    THError("MAGMA gels : Argument %d : illegal value", -info);

  THZCudaTensor_freeCopyTo(state, a, ra_);
  THZCudaTensor_freeCopyTo(state, b, rb_);
#else
  THError(NoMagma(gels));
#endif
}

void THZCudaTensor_syev(THCState *state, THZCudaTensor *re_, THZCudaTensor *rv_, THZCudaTensor *a, const char *jobzs, const char *uplos)
{
#ifdef USE_MAGMA
  int n = a->size[0];
  int lda = n;

  magma_uplo_t uplo = uplos[0] == 'U' ?  MagmaUpper : MagmaLower;
  magma_vec_t jobz = jobzs[0] == 'N' ? MagmaNoVec : MagmaVec;

  THZCudaTensor *input = THZCudaTensor_newColumnMajor(state, rv_, a);
  float *input_data = THZCudaTensor_data(state, input);

  // eigen values and workspace
  float *w = th_magma_smalloc_pinned(n);
  float *wA = th_magma_smalloc_pinned(lda);

  // compute optimal size of work array
  int info;
  float lwork;
  int liwork;
  magma_ssyevd_gpu(jobz, uplo, n, input_data, lda, w, wA, n, &lwork, -1, &liwork, -1, &info);

  float *work = th_magma_smalloc_pinned((size_t)lwork);
  int *iwork = th_magma_imalloc_pinned(liwork);

  // compute eigenvalues and, optionally, eigenvectors
  magma_ssyevd_gpu(jobz, uplo, n, input_data, lda, w, wA, n, work, (int) lwork, iwork, liwork, &info);

  // copy eigen values from w to re_
  if (info == 0)
    THZCudaTensor_copyArray1d(state, re_, w, n);

  magma_free_pinned(iwork);
  magma_free_pinned(work);
  magma_free_pinned(wA);
  magma_free_pinned(w);

  // check error value
  if (info > 0)
    THError("MAGMA syev : Failed to converge. %d off-diagonal elements of an didn't converge to zero", info);
  else if (info < 0)
    THError("MAGMA syev : Argument %d : illegal value", -info);

  THZCudaTensor_freeCopyTo(state, input, rv_);
#else
  THError(NoMagma(syev));
#endif
}

void THZCudaTensor_geev(THCState *state, THZCudaTensor *re_, THZCudaTensor *rv_, THZCudaTensor *a_, const char *jobvrs)
{
#ifdef USE_MAGMA
  THArgCheck(a_->nDimension == 2, 3, "A should be 2 dimensional");
  THArgCheck(a_->size[0] == a_->size[1], 3, "A should be square");

  magma_vec_t jobvr = jobvrs[0] == 'N' ? MagmaNoVec : MagmaVec;
  int n = a_->size[0];

  float *a_data = th_magma_smalloc_pinned(n * n);
  THZCudaTensor_copyTensor2d(state, a_data, a_);

  float *wr = th_magma_smalloc_pinned(n);
  float *wi = th_magma_smalloc_pinned(n);

  float *vr_data = NULL;
  int ldvr = 1;
  if (jobvr == MagmaVec)
  {
    vr_data = th_magma_smalloc_pinned(n * n);
    ldvr = n;
  }

  float wkopt;
  int info;

  magma_sgeev(MagmaNoVec, jobvr, n, a_data, n, wr, wi, NULL, 1, vr_data, ldvr, &wkopt, -1, &info);

  int lwork = (int) wkopt;
  float *work_data = th_magma_smalloc_pinned(lwork);

  magma_sgeev(MagmaNoVec, jobvr, n, a_data, n, wr, wi, NULL, 1, vr_data, ldvr, work_data, lwork, &info);

  if (info > 0)
    THError("MAGMA geev : Failed to converge. %d off-diagonal elements of an didn't converge to zero", info);
  else if (info < 0)
    THError("MAGMA geev : Argument %d : illegal value", -info);

  {
    THZCudaTensor_resize2d(state, re_, 2, n);
    THZCudaTensor *re = THZCudaTensor_newContiguous(state, re_);
    THZCudaCheck(hipMemcpy(re->storage->data + re->storageOffset, wr, n*sizeof(float), hipMemcpyHostToDevice));
    THZCudaCheck(hipMemcpy(re->storage->data + re->storageOffset + n, wi, n*sizeof(float), hipMemcpyHostToDevice));
    THZCudaTensor_freeCopyTo(state, re, re_);
    THZCudaTensor_transpose(state, re_, NULL, 0, 1);
  }

  if (jobvr == MagmaVec)
    THZCudaTensor_copyArray2d(state, rv_, vr_data, n, n);

  magma_free_pinned(work_data);
  magma_free_pinned(vr_data);
  magma_free_pinned(wi);
  magma_free_pinned(wr);
  magma_free_pinned(a_data);

#else
  THError(NoMagma(geev));
#endif
}

void THZCudaTensor_gesvd(THCState *state, THZCudaTensor *ru_, THZCudaTensor *rs_, THZCudaTensor *rv_, THZCudaTensor *a, const char *jobu)
{
#ifdef USE_MAGMA
  THZCudaTensor *ra_ = THZCudaTensor_new(state);
  THZCudaTensor_gesvd2(state, ru_, rs_, rv_,  ra_, a, jobu);
  THZCudaTensor_free(state, ra_);
#else
  THError(NoMagma(gesvd));
#endif
}

void THZCudaTensor_gesvd2(THCState *state, THZCudaTensor *ru_, THZCudaTensor *rs_, THZCudaTensor *rv_, THZCudaTensor *ra_, THZCudaTensor *a, const char *jobus)
{
#ifdef USE_MAGMA
  THArgCheck(a->nDimension == 2, 2, "A should be 2 dimensional");

  magma_vec_t jobu = jobus[0] == 'A' ? MagmaAllVec : jobus[0] == 'S' ? MagmaSomeVec : jobus[0] == 'O' ? MagmaOverwriteVec : MagmaNoVec;
  magma_vec_t jobvt = jobu;

  int m = a->size[0];
  int n = a->size[1];
  int k = m < n ? m : n;
  int j = (jobu == MagmaAllVec) ? m : k;

  float *a_data = th_magma_smalloc_pinned(m * n);
  THZCudaTensor_copyTensor2d(state, a_data, a);

  float *rs_data = th_magma_smalloc_pinned(k);
  float *ru_data = th_magma_smalloc_pinned(m * j);
  float *rv_data = th_magma_smalloc_pinned(n * n);

  float wkopt;
  int info;
  magma_sgesvd(jobu, jobvt, m, n, a_data, m, rs_data, ru_data, m, rv_data, n, &wkopt, -1, &info);

  int lwork = (int) wkopt;
  float *work_data = th_magma_smalloc_pinned(lwork);

  magma_sgesvd(jobu, jobvt, m, n, a_data, m, rs_data, ru_data, m, rv_data, n, work_data, lwork, &info);

  if (info > 0)
    THError("MAGMA gesvd : %d superdiagonals failed to converge", info);
  else if (info < 0)
    THError("MAGMA gesvd : Argument %d : illegal value", -info);

  THZCudaTensor_copyArray2d(state, rv_, rv_data, n, n);
  THZCudaTensor_transpose(state, rv_, NULL, 0, 1);
  THZCudaTensor_copyArray2d(state, ru_, ru_data, m, j);
  THZCudaTensor_copyArray1d(state, rs_, rs_data, k);
  THZCudaTensor_copyArray2d(state, ra_, a_data,  m, n);

  magma_free_pinned(work_data);
  magma_free_pinned(rv_data);
  magma_free_pinned(ru_data);
  magma_free_pinned(rs_data);
  magma_free_pinned(a_data);
#else
  THError(NoMagma(gesvd2));
#endif
}

void THZCudaTensor_getri(THCState *state, THZCudaTensor *ra_, THZCudaTensor *a)
{
#ifdef USE_MAGMA
  THArgCheck(a->nDimension == 2, 2, "A should be 2 dimensional");
  THArgCheck(a->size[0] == a->size[1], 2, "A should be square");

  int info;
  int n = a->size[0];
  int lwork = n * magma_get_sgetri_nb(n);

  THZCudaTensor *input = THZCudaTensor_newColumnMajor(state, ra_, a);
  float *input_data = THZCudaTensor_data(state, input);

  int *ipiv = th_magma_imalloc_pinned(n);

  THZCudaTensor *work = THZCudaTensor_newWithSize1d(state, lwork);
  float *work_data = THZCudaTensor_data(state, work);

  // Run LU
  magma_sgetrf_gpu(n, n, input_data, n, ipiv, &info);
  if (info > 0)
    THError("MAGMA getrf : U(%d,%d) is 0, U is singular", info, info);
  else if (info < 0)
    THError("MAGMA getrf : Argument %d : illegal value", -info);

  // Inverse
  magma_sgetri_gpu(n, input_data, n, ipiv, work_data, lwork, &info);
  if (info > 0)
    THError("MAGMA getri : U(%d,%d) is 0, U is singular", info, info);
  else if (info < 0)
    THError("MAGMA getri : Argument %d : illegal value", -info);

  THZCudaTensor_free(state, work);
  magma_free_pinned(ipiv);
  THZCudaTensor_freeCopyTo(state, input, ra_);
#else
  THError(NoMagma(getri));
#endif
}

__global__ void THZCudaTensor_copyUpperSymmetric(float *input, int n, int len)
{
  for (int idx = threadIdx.x + blockIdx.x * blockDim.x; idx < len; idx += 65535) {
    const int r = idx % n;
    const int c = idx / n;
    if (r > c) {
      input[idx] = input[r*n + c];
    }
  }
}

void THZCudaTensor_potri(THCState *state, THZCudaTensor *ra_, THZCudaTensor *a)
{
#ifdef USE_MAGMA
  THArgCheck(a->nDimension == 2, 2, "A should be 2 dimensional");
  THArgCheck(a->size[0] == a->size[1], 2, "A should be square");

  int n = a->size[0];

  THZCudaTensor *input = THZCudaTensor_newColumnMajor(state, ra_, a);
  float *input_data = THZCudaTensor_data(state, input);

  int info;
  magma_spotrf_gpu(MagmaUpper, n, input_data, n, &info);
  if (info > 0)
    THError("MAGMA potrf : A(%d,%d) is 0, A cannot be factorized", info, info);
  else if (info < 0)
    THError("MAGMA potrf : Argument %d : illegal value", -info);

  magma_spotri_gpu(MagmaUpper, n, input_data, n, &info);
  if (info > 0)
    THError("MAGMA potri : A(%d,%d) is 0, A cannot be factorized", info, info);
  else if (info < 0)
    THError("MAGMA potri : Argument %d : illegal value", -info);

  hipStream_t stream = THCState_getCurrentStream(state);
  const int len = n*n;
  dim3 blocks(std::min(DIVUP(len, 128), 65535));
  dim3 threads(128);
  THZCudaTensor_copyUpperSymmetric<<<blocks, threads, 0, stream>>>(input_data, n, len);

  THZCudaTensor_freeCopyTo(state, input, ra_);
#else
  THError(NoMagma(potri));
#endif
}

void THZCudaTensor_potrf(THCState *state, THZCudaTensor *ra_, THZCudaTensor *a)
{
#ifdef USE_MAGMA
  THArgCheck(a->nDimension == 2, 2, "A should be 2 dimensional");
  THArgCheck(a->size[0] == a->size[1], 2, "A should be square");

  int n = a->size[0];

  THZCudaTensor *input = THZCudaTensor_newColumnMajor(state, ra_, a);
  float *input_data = THZCudaTensor_data(state, input);

  int info;
  magma_spotrf_gpu(MagmaUpper, n, input_data, n, &info);

  // check error value
  if (info > 0)
    THError("MAGMA potrf : A(%d,%d) is 0, A cannot be factorized", info, info);
  else if (info < 0)
    THError("MAGMA potrf : Argument %d : illegal value", -info);

  THZCudaTensor_triu(state, ra_, input, 0);
  THZCudaTensor_free(state, input);
#else
  THError(NoMagma(potrf));
#endif
}

void THZCudaTensor_qr(THCState *state, THZCudaTensor *rq_, THZCudaTensor *rr_, THZCudaTensor *a_)
{
#ifdef USE_MAGMA
  THArgCheck(a_->nDimension == 2, 2, "A should be 2 dimensional");

  THZCudaTensor *a = THZCudaTensor_newColumnMajor(state, rr_, a_);
  int m = a->size[0];
  int n = a->size[1];
  int k = (m < n ? m : n);
  int nb = magma_get_sgeqrf_nb(m);

  float *a_data = THZCudaTensor_data(state, a);
  float *tau_data = th_magma_smalloc_pinned(n*n);

  THZCudaTensor *work = THZCudaTensor_newWithSize1d(state, (2*k + ((n+31)/32)*32)*nb);
  float *work_data = THZCudaTensor_data(state, work);

  int info;
  magma_sgeqrf_gpu(m, n, a_data, m, tau_data, work_data, &info);

  if (info != 0)
    THError("MAGMA geqrf : Argument %d : illegal value.", -info);

  THZCudaTensor *q = THZCudaTensor_newColumnMajor(state, rq_, a);
  float *q_data = THZCudaTensor_data(state, q);

  THZCudaTensor_narrow(state, a, a, 0, 0, k);
  THZCudaTensor_triu(state, rr_, a, 0);
  THZCudaTensor_free(state, a);

  magma_sorgqr_gpu(m, n, k, q_data, m, tau_data, work_data, nb, &info);

  if (info != 0)
    THError("MAGMA orgqr : Argument %d : illegal value.", -info);

  THZCudaTensor_free(state, work);
  magma_free_pinned(tau_data);

  THZCudaTensor_narrow(state, q, q, 1, 0, k);
  THZCudaTensor_freeCopyTo(state, q, rq_);
#else
  THError(NoMagma(qr));
#endif
}
