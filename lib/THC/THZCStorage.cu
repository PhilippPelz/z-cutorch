#include "THZCStorage.h"

#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#if CUDA_VERSION >= 7000
#include <thrust/system/cuda/execution_policy.h>
#endif

void THZCudaStorage_fill(THCState *state, THZCudaStorage *self, float value)
{
  thrust::device_ptr<float> self_data(self->data);
  thrust::fill(
#if CUDA_VERSION >= 7000
    thrust::cuda::par.on(THCState_getCurrentStream(state)),
#endif
    self_data, self_data+self->size, value);
}

void THZCudaStorage_resize(THCState *state, THZCudaStorage *self, long size)
{
  THArgCheck(size >= 0, 2, "invalid size");

  if(!(self->flag & TH_STORAGE_RESIZABLE))
    return;

  if(size == 0)
  {
    if(self->flag & TH_STORAGE_FREEMEM) {
      THZCudaCheck(THZCudaFree(state, self->data));
      THZCHeapUpdate(state, -self->size * sizeof(float));
    }
    self->data = NULL;
    self->size = 0;
  }
  else
  {
    float *data = NULL;
    // update heap *before* attempting malloc, to free space for the malloc
    THZCHeapUpdate(state, size * sizeof(float));
    hipError_t err = THZCudaMalloc(state, (void**)(&data), size * sizeof(float));
    if(err != hipSuccess) {
      THZCHeapUpdate(state, -size * sizeof(float));
    }
    THZCudaCheck(err);

    if (self->data) {
      THZCudaCheck(hipMemcpyAsync(data,
                                  self->data,
                                  THMin(self->size, size) * sizeof(float),
                                  hipMemcpyDeviceToDevice,
                                  THCState_getCurrentStream(state)));
      THZCudaCheck(THZCudaFree(state, self->data));
      THZCHeapUpdate(state, -self->size * sizeof(float));
    }

    self->data = data;
    self->size = size;
  }
}
