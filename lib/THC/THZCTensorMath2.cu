#include "hip/hip_runtime.h"
#include "THZCTensorMath.h"
// #include "THZCGeneral.h"
#include "THZCBlas.h"
#include "THZCTensorCopy.h"
// #include "THZCTensorRandom.h"
#include "THZCApply.cuh"
#include "THZCReduce.cuh"

#include <thrust/device_ptr.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/inner_product.h>

// #include <thrust/complex.h>
// typedef thrust::complex<float> ccx;

#if CUDA_VERSION >= 7000
#include <thrust/system/cuda/execution_policy.h>
#endif

// ccx toCcx(cx val) {
// 	return ccx(crealf(val), cimagf(val));
// }

struct ZTensorPowOp {
  ZTensorPowOp(ccx v) : val(v) {}
  __device__ __forceinline__ void operator()(ccx* out, ccx* in) {
    *out = thrust::pow(*in, val);
  }

  __device__ __forceinline__ void operator()(ccx* v) {
    *v = thrust::pow(*v, val);
  }

  const ccx val;
};

void THZCudaTensor_pow(THCState *state, THZCudaTensor *self_, THZCudaTensor *src, cx value)
{
  THAssert(THZCudaTensor_checkGPU(state, 2, self_, src));
  if (self_ == src) {
    if (!THZCudaTensor_pointwiseApply1(state, self_, ZTensorPowOp(toCcx(value)))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THZCudaTensor_resizeAs(state, self_, src);

    if (!THZCudaTensor_pointwiseApply2(state, self_, src, ZTensorPowOp(toCcx(value)))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THZCudaCheck(hipGetLastError());
}

cx THZCudaTensor_meanall(THCState *state, THZCudaTensor *self)
{
  THAssert(THZCudaTensor_checkGPU(state, 1, self));
  THArgCheck(self->nDimension > 0, 1, "empty Tensor");
  return THZCudaTensor_sumall(state, self)/THZCudaTensor_nElement(state, self);
}

void
THZCudaTensor_mean(THCState *state, THZCudaTensor *self, THZCudaTensor *src, long dim)
{
  THAssert(THZCudaTensor_checkGPU(state, 2, self, src));
  THZCudaTensor_sum(state, self, src, dim);
  THZCudaTensor_div(state, self, self, THZCudaTensor_size(state, src, dim));
}

struct square_functor
{
  const ccx mean;

  square_functor(ccx mean_) : mean(mean) {}

    __host__ __device__ ccx operator()(const ccx& x) const
  {
    float x1 = thrust::abs((ccx)x-mean);
    return ccx(x1*x1,0);
  }
};

float THZCudaTensor_varall(THCState *state, THZCudaTensor *self)
{
  THAssert(THZCudaTensor_checkGPU(state, 1, self));
  self = THZCudaTensor_newContiguous(state, self);
  long size = THZCudaTensor_nElement(state, self);
  thrust::device_ptr<ccx> self_data((ccx*)THZCudaTensor_data(state, self));

  cx mean = THZCudaTensor_meanall(state, self);
  ccx result =
    thrust::transform_reduce(
#if CUDA_VERSION >= 7000
      thrust::cuda::par.on(THCState_getCurrentStream(state)),
#endif
      self_data, self_data+size, square_functor(toCcx(mean)),
      ccx(0,0), thrust::plus<ccx>());

  float res = result.real()/(float)(THZCudaTensor_nElement(state, self)-1);

  THZCudaTensor_free(state, self);
  return res;
}

float THZCudaTensor_stdall(THCState *state, THZCudaTensor *self)
{
  THAssert(THZCudaTensor_checkGPU(state, 1, self));
  return sqrt(THZCudaTensor_varall(state, self));
}

template <int StaticExp>
struct TensorNormOp
{
  TensorNormOp(float exp) : exponent(exp) {}

  __host__ __device__ ccx operator()(ccx y) const {
    if (StaticExp == 1) {
      return ccx(thrust::abs(y),0);
    } else if (StaticExp == 2) {
      float x = thrust::abs(y);
      return ccx(x * x,0);
    } else {
      return ccx(powf(thrust::abs(y), exponent),0);
    }
  }

  const float exponent;
};

struct TensorNonZeroOp
{
  TensorNonZeroOp() {}
  __host__ __device__ ccx operator()(ccx lhs) const { return thrust::abs(lhs) != 0.0f ? ccx(1,0) : ccx(0,0); }
};

float THZCudaTensor_normall(THCState *state, THZCudaTensor *self, float value)
{
  THAssert(THZCudaTensor_checkGPU(state, 1, self));
  self = THZCudaTensor_newContiguous(state, self);
  long size = THZCudaTensor_nElement(state, self);
  thrust::device_ptr<ccx> self_data((ccx*)THZCudaTensor_data(state, self));

  ccx result;
	float res = 0;
  if (value == 0.0f) {
    result = thrust::transform_reduce(
#if CUDA_VERSION >= 7000
      thrust::cuda::par.on(THCState_getCurrentStream(state)),
#endif
      self_data, self_data+size, TensorNonZeroOp(),
      ccx(0.0,0), thrust::plus<ccx>());
			res = result.real();
  } else if (value == 1.0f) {
    result = thrust::transform_reduce(
#if CUDA_VERSION >= 7000
      thrust::cuda::par.on(THCState_getCurrentStream(state)),
#endif
      self_data, self_data+size, TensorNormOp<1>(value),
      ccx(0.0,0), thrust::plus<ccx>());

  } else if (value == 2.0f) {
    result = thrust::transform_reduce(
#if CUDA_VERSION >= 7000
      thrust::cuda::par.on(THCState_getCurrentStream(state)),
#endif
      self_data, self_data+size, TensorNormOp<2>(value),
      ccx(0.0,0), thrust::plus<ccx>());
    res = powf(result.real(), 0.5f);

  } else {
    result = thrust::transform_reduce(
#if CUDA_VERSION >= 7000
      thrust::cuda::par.on(THCState_getCurrentStream(state)),
#endif
      self_data, self_data+size, TensorNormOp<-1>(value),
      ccx(0.0,0), thrust::plus<ccx>());
    res = powf(result.real(), 1.0f / value);
  }

  THZCudaTensor_free(state, self);
  return res;
}

void THZCudaTensor_normDim(THCState *state, THZCudaTensor* self, THZCudaTensor* src, float value, long dimension)
{
  THAssert(THZCudaTensor_checkGPU(state, 2, self, src));
  if (value == 0.0f) {
    THZCudaTensor_reduceDim(state, self, src,
                           TensorNonZeroOp(), thrust::plus<ccx>(),
                           0.0f, dimension);
  } else if (value == 1.0f) {
    THZCudaTensor_reduceDim(state, self, src,
                           TensorNormOp<1>(value), thrust::plus<ccx>(),
                           0.0f, dimension);

  } else if (value == 2.0f) {
    THZCudaTensor_reduceDim(state, self, src,
                           TensorNormOp<2>(value), thrust::plus<ccx>(),
                           0.0f, dimension);
    THZCudaTensor_pow(state, self, self, 0.5f);

  } else {
    THZCudaTensor_reduceDim(state, self, src,
                           TensorNormOp<-1>(value), thrust::plus<ccx>(),
                           0.0f, dimension);
    THZCudaTensor_pow(state, self, self, 1.0f / value);
  }

  THZCudaCheck(hipGetLastError());
}

__global__ void THZCudaTensor_kernel_renorm(ccx *data, const float value, const long size, const float maxnorm)
{
  __shared__ float buffer[32];
  long tx = threadIdx.x;
  long bx = blockIdx.x;
  long step = blockDim.x;
  ccx *row = data + size*bx;

  buffer[tx] = 0;

  // get norm of axis
  for (long i=tx; i<size; i+=step)
  {
    buffer[tx] += powf(thrust::abs(row[i]), value);
  }
  // add (reduce)
  for (unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1)
  {
    __syncthreads();
    if (tx < stride)
      buffer[tx] += buffer[tx+stride];
  }
  // clip norms
  __syncthreads();
  float norm = powf(buffer[0], 1/value);
  if (norm > maxnorm)
  {
    norm = maxnorm / (norm + 1e-7);
    // renormalize
    for (long i=tx; i<size; i+=step)
    {
      row[i] *= norm;
    }
  }
}

void THZCudaTensor_renorm(THCState *state, THZCudaTensor* self, THZCudaTensor* src, float value, long dimension, float maxnorm)
{
  THAssert(THZCudaTensor_checkGPU(state, 2, self, src));
  THZCudaTensor *self_;
  THZCudaTensor *src_ = THZCudaTensor_newTranspose(state, src, dimension, 0);
  THZCudaTensor *data = THZCudaTensor_newClone(state, src_);
  long size = THZCudaTensor_nElement(state, data)/data->size[0];

  THArgCheck(dimension >= 0 && dimension < THZCudaTensor_nDimension(state, src), 3, "invalid dimension");
  THArgCheck(value > 0, 2, "non-positive-norm not supported");
  THArgCheck(THZCudaTensor_nDimension(state, src) > 1, 1, "need at least 2 dimensions");

  dim3 grid(data->size[0]);
  dim3 threads(32);

  THZCudaTensor_kernel_renorm<<<grid, threads, 0, THCState_getCurrentStream(state)>>>((ccx*)THZCudaTensor_data(state, data), value, size, maxnorm);

  hipError_t errcode = hipGetLastError();
  if(errcode != hipSuccess)
    THError(hipGetErrorString(errcode));

  THZCudaTensor_free(state, src_);
  self_ = THZCudaTensor_newTranspose(state, data, dimension, 0);
  THZCudaTensor_resizeAs(state, self, self_);
  THZCudaTensor_freeCopyTo(state, self_, self);
  THZCudaTensor_free(state, data);
}

struct dist_functor
{
  const float exponent;

  dist_functor(float exponent_) : exponent(exponent_) {}

  __host__ __device__ ccx operator()(const ccx& x, const ccx& y) const
  {
    return ccx(powf(thrust::abs(x-y), exponent),0);
  }
};

float THZCudaTensor_dist(THCState *state, THZCudaTensor *self, THZCudaTensor *src, float value)
{
  THAssert(THZCudaTensor_checkGPU(state, 2, self, src));
  self = THZCudaTensor_newContiguous(state, self);
  long size = THZCudaTensor_nElement(state, self);
  src = THZCudaTensor_newContiguous(state, src);
  thrust::device_ptr<ccx> self_data((ccx*)THZCudaTensor_data(state, self));
  thrust::device_ptr<ccx> src_data((ccx*)THZCudaTensor_data(state, src));
 ccx result;
//   ccx result = thrust::inner_product(
// #if CUDA_VERSION >= 7000
//     thrust::cuda::par.on(THCState_getCurrentStream(state)),
// #endif
//     self_data, self_data+size, src_data, ccx(0,0),
//     thrust::plus<ccx>, dist_functor(value));

  THZCudaTensor_free(state, src);
  THZCudaTensor_free(state, self);

  return powf(result.real(), (float)1.0/value);
}
